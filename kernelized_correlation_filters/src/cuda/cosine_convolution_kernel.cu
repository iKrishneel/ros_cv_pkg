#include "hip/hip_runtime.h"

#include <kernelized_correlation_filters/cosine_convolution_kernel.h>

__host__ __forceinline__
void cuAssert(hipError_t code, char *file, int line, bool abort) {
    if (code != hipSuccess) {
       fprintf(stderr, "GPUassert: %s %s %dn",
               hipGetErrorString(code), file, line);
       if (abort) {
          exit(code);
      }
    }
}


__host__ __device__ __align__(16)
int cuDivUp(int a, int b) {
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


__global__ __forceinline__
void cosineConvolutionKernel(float *d_output,
                             const float*d_cnn_codes,
                             const float *d_cos_window,
                             const int data_count) {
   
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < data_count) {
       d_output[offset] = d_cnn_codes[offset] * d_cos_window[offset];

       /*
       if (offset > 109 && offset < 120) {
          printf("%3.4f  %3.4f   %3.4f \n", d_output[offset],
                 d_cnn_codes[offset], d_cos_window[offset]);
       }
       */
    }
}


float* cosineConvolutionGPU(const float *d_cnn_codes,
                            const float *d_cos_window,
                            const int data_count,
                            const int BYTE) {

     const int dimension = std::ceil(std::sqrt(data_count));
     dim3 grid_size(cuDivUp(dimension, GRID_SIZE),
                     cuDivUp(dimension, GRID_SIZE));
     dim3 block_size(GRID_SIZE, GRID_SIZE);
     
     float *d_output;
     hipMalloc(reinterpret_cast<void**>(&d_output), BYTE);
     cosineConvolutionKernel<<<grid_size, block_size>>>(
        d_output, d_cnn_codes, d_cos_window, data_count);

     // printf("GPU : %d %d\n", dimension, data_count);
     
     return d_output;
}
