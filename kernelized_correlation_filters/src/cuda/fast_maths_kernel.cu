#include "hip/hip_runtime.h"

/**
 * adapted from NVIDIA hip/hip_complex.h
 */

#include <kernelized_correlation_filters/fast_maths_kernel.h>

/**
 * multiplication
 */

__global__ __forceinline__
void multiplyComplexKernel(hipfftComplex *d_results,
                           const hipfftComplex *d_complex1,
                           const hipfftComplex *d_complex2,
                           const int dimension) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < dimension) {
       d_results[offset].x = (d_complex1[offset].x * d_complex2[offset].x) -
          (d_complex1[offset].y * d_complex2[offset].y);
       d_results[offset].y = (d_complex1[offset].x * d_complex2[offset].y) +
          (d_complex1[offset].y * d_complex2[offset].x);
    }
}

// (a+bi)(c+di) = (a*c - b*d), (a*d+c*b)
hipfftComplex* multiplyComplexGPU(const hipfftComplex *d_complex1,
                                 const hipfftComplex *d_complex2,
                                 const int dimension) {
    if (dimension == 0) {
       printf("ERROR: [multiplyComplexGPU] DATA DIMENSION = 0\n");
       hipfftComplex empty[1];
       return empty;
    }
    
    const int csize = std::ceil(std::sqrt(dimension));
    dim3 grid_size(cuDivUp(csize, GRID_SIZE),
                   cuDivUp(csize, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    int BYTE = dimension * sizeof(hipfftComplex);
    hipfftComplex *d_results;
    hipMalloc(reinterpret_cast<void**>(&d_results), BYTE);
    multiplyComplexKernel<<<grid_size, block_size>>>(d_results, d_complex1,
                                                     d_complex2, dimension);

    return d_results;
}


/**
 * mulitply by scalar
 */

__global__ __forceinline__
void multiplyComplexByScalarKernel(hipfftComplex *d_results,
                                   const hipfftComplex *d_complex,
                                   const hipfftComplex scalar,
                                   const int dimension) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < dimension) {
       d_results[offset].x =
          (d_complex[offset].x * scalar.x) -
          (d_complex[offset].y * scalar.y);
       d_results[offset].y =
          (d_complex[offset].x * scalar.y) +
          (d_complex[offset].y * scalar.x);
    }
}

hipfftComplex* multiplyComplexByScalarGPU(const hipfftComplex *d_complex,
                                         const float scalar,
                                         const int dimension) {
    if (dimension == 0) {
       printf("ERROR: [multiplyComplexByScalarGPU] DATA DIMENSION = 0\n");
       hipfftComplex empty[1];
       return empty;
    }
    const int csize = std::ceil(std::sqrt(dimension));
    dim3 grid_size(cuDivUp(csize, GRID_SIZE),
                   cuDivUp(csize, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    int BYTE = dimension * sizeof(hipfftComplex);
    hipfftComplex *d_results;
    hipMalloc(reinterpret_cast<void**>(&d_results), BYTE);

    hipfftComplex scalar_complex;
    scalar_complex.x = scalar;
    scalar_complex.y = 0.0f;

    multiplyComplexByScalarKernel<<<grid_size, block_size>>>(
       d_results, d_complex, scalar_complex, dimension);
    return d_results;
}

/**
 * addition kernel
 */
__global__ __forceinline__
void addComplexKernel(hipfftComplex *d_results,
                      const hipfftComplex *d_complex1,
                      const hipfftComplex *d_complex2,
                      const int dimension) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < dimension) {
       d_results[offset].x = d_complex1[offset].x + d_complex2[offset].x;
       d_results[offset].y = d_complex1[offset].y + d_complex2[offset].y;
    }
}

hipfftComplex* addComplexGPU(const hipfftComplex *d_complex1,
                            const hipfftComplex *d_complex2,
                            const int dimension) {
    if (dimension == 0) {
       printf("ERROR: [addComplexGPU] DATA DIMENSION = 0\n");
       hipfftComplex empty[1];
       return empty;
    }
    const int csize = std::ceil(std::sqrt(dimension));
    dim3 grid_size(cuDivUp(csize, GRID_SIZE),
                   cuDivUp(csize, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    int BYTE = dimension * sizeof(hipfftComplex);
    hipfftComplex *d_results;
    hipMalloc(reinterpret_cast<void**>(&d_results), BYTE);
    addComplexKernel<<<grid_size, block_size>>>(d_results, d_complex1,
                                                d_complex2, dimension);
    return d_results;
}

/**
 * add by scalar
 */

__global__ __forceinline__
void addComplexByScalarKernel(hipfftComplex *d_results,
                              const hipfftComplex *d_complex,
                              const float scalar,
                              const int dimension) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < dimension) {
       d_results[offset] = d_complex[offset];
       d_results[offset].x = d_complex[offset].x + scalar;
    }
}

hipfftComplex* addComplexByScalarGPU(const hipfftComplex *d_complex,
                                    const float scalar,
                                    const int dimension) {
    if (dimension == 0) {
       printf("ERROR: [addComplexByScalarGPU] DATA DIMENSION = 0\n");
       hipfftComplex empty[1];
       return empty;
    }
    const int csize = std::ceil(std::sqrt(dimension));
    dim3 grid_size(cuDivUp(csize, GRID_SIZE),
                   cuDivUp(csize, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);
    
    int BYTE = dimension * sizeof(hipfftComplex);
    hipfftComplex *d_results;
    hipMalloc(reinterpret_cast<void**>(&d_results), BYTE);
    addComplexByScalarKernel<<<grid_size, block_size>>>(d_results, d_complex,
                                                        scalar, dimension);
    return d_results;
}

/**
 * division
 */

__global__ __forceinline__
void divisionComplexKernel(hipfftComplex *d_results,
                           const hipfftComplex *d_complex1,
                           const hipfftComplex *d_complex2,
                           const int dimension) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < dimension) {

       float s = fabsf(d_complex2[offset].x) + fabsf(d_complex2[offset].y);
       float oos = 1.0f / s;
       float ars = d_complex1[offset].x * oos;
       float ais = d_complex1[offset].y * oos;
       float brs = d_complex2[offset].x * oos;
       float bis = d_complex2[offset].y * oos;
       s = (brs * brs) + (bis * bis);
       oos = 1.0f / s;

       d_results[offset].x = ((ars * brs) + (ais * bis)) * oos;
       d_results[offset].y = ((ais * brs) - (ars * bis)) * oos;
    }
}

hipfftComplex* divisionComplexGPU(const hipfftComplex *d_complex1,
                                 const hipfftComplex *d_complex2,
                                 const int dimension) {
    if (dimension == 0) {
       printf("ERROR: [addComplexGPU] DATA DIMENSION = 0\n");
       hipfftComplex empty[1];
       return empty;
    }
    const int csize = std::ceil(std::sqrt(dimension));
    dim3 grid_size(cuDivUp(csize, GRID_SIZE),
                   cuDivUp(csize, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    int BYTE = dimension * sizeof(hipfftComplex);
    hipfftComplex *d_results;
    hipMalloc(reinterpret_cast<void**>(&d_results), BYTE);
    
    divisionComplexKernel<<<grid_size, block_size>>>(d_results, d_complex1,
                                                d_complex2, dimension);
    return d_results;
}
