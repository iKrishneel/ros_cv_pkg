#include "hip/hip_runtime.h"

#include <kernelized_correlation_filters/gaussian_correlation_kernel.h>


__device__ __forceinline__
float squaredMagnitude(const hipfftComplex data) {
    return (powf(data.x, 2) + powf(data.y, 2));
}

__global__ __forceinline__
void squaredNormKernel(float *d_squared_norm,
                       const hipfftComplex *d_complex,
                       const int LENGHT) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < LENGHT) {
       // d_squared_norm[offset] = squaredMagnitude(d_complex[offset]);
       d_squared_norm[offset] = (d_complex[offset].x * d_complex[offset].x) +
          (d_complex[offset].y * d_complex[offset].y);

       /*
       if (isnan(d_squared_norm[offset])) {
          printf("GPU DEBUG: %d  %3.5f  %3.5f\n", offset,
                 d_complex[offset].x, d_complex[offset].y);
       }
       */
    }
}



float squaredNormGPU(const hipfftComplex *d_complex,
                     const int FILTER_BATCH,
                     const int FILTER_SIZE) {
    if (FILTER_BATCH == 0 || FILTER_SIZE == 0) {
       printf("\033[31m ERROR: [squaredNormGPU] FAILED\n");
    }
    int LENGHT = FILTER_BATCH * FILTER_SIZE;
    
    float *d_squared_norm;
    const int BYTE = LENGHT * sizeof(float);
    hipMalloc(reinterpret_cast<void**>(&d_squared_norm), BYTE);

    const int dimension = std::ceil(std::sqrt(LENGHT));
    dim3 grid_size(cuDivUp(dimension, GRID_SIZE),
                   cuDivUp(dimension, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);
    squaredNormKernel<<<grid_size, block_size>>>(d_squared_norm,
                                                 d_complex, LENGHT);
    
    float *d_summation;
    hipMalloc(reinterpret_cast<void**>(&d_summation), BYTE);

    // TODO(TX1):  check and set auto
    int num_threads = 128;
    int num_blocks = 64;

    reduceSinglePass(LENGHT, num_threads, num_blocks,
                     d_squared_norm, d_summation);

    float *sum = reinterpret_cast<float*>(std::malloc(BYTE));
    hipMemcpy(sum, d_summation, BYTE, hipMemcpyDeviceToHost);
    
    float norm = sum[0] / FILTER_SIZE;
    
    free(sum);
    hipFree(d_squared_norm);
    hipFree(d_summation);

    return norm;
}


/**
 * kernel for computing just the inverse
 */
__global__ __forceinline__
void invComplexConjuateKernel(hipfftComplex *d_compl_out,
                              const hipfftComplex *d_complex,
                              const int LENGHT) {

    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < LENGHT) {
       d_compl_out[offset] = d_complex[offset];
       d_compl_out[offset].y *= -1.0f;
    }
   
}

hipfftComplex* invComplexConjuateGPU(const hipfftComplex *d_complex,
                                    const int FILTER_BATCH,
                                    const int FILTER_SIZE) {

    if (FILTER_BATCH == 0 || FILTER_SIZE == 0) {
       printf("\033[31m ERROR: [invComplexConjuateGPU] FAILED\n");
    }
    
    int LENGHT = FILTER_BATCH * FILTER_SIZE;
    const int dimension = std::ceil(std::sqrt(LENGHT));
    dim3 grid_size(cuDivUp(dimension, GRID_SIZE),
                   cuDivUp(dimension, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    const int BYTE = LENGHT * sizeof(hipfftComplex);
    hipfftComplex *d_compl_out;
    hipMalloc(reinterpret_cast<void**>(&d_compl_out), BYTE);

    invComplexConjuateKernel<<<grid_size, block_size>>>(d_compl_out,
                                                        d_complex, LENGHT);
    
    return d_compl_out;
}


/**
 * kernel to inverse and multipy reduced into one
 */

__global__ __forceinline__
void invConjuateConvKernel(hipfftComplex *d_compl_out,
                           const hipfftComplex *d_complex,
                           const hipfftComplex *d_compl_model,
                           const int LENGHT) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (offset < LENGHT) {
       d_compl_out[offset].x = (
          (d_complex[offset].x * d_compl_model[offset].x) -
          (d_complex[offset].y * (d_compl_model[offset].y * -1.0f)));
       d_compl_out[offset].y = 0.0f;
    }
}

hipfftComplex* invConjuateConvGPU(const hipfftComplex *d_complex,
                                 const hipfftComplex *d_compl_model,
                                 const int FILTER_BATCH,
                                 const int FILTER_SIZE) {

    if (FILTER_BATCH == 0 || FILTER_SIZE == 0) {
       printf("\033[31m ERROR: [invConjuateConvGPU] FAILED\n");
    }
    
    int LENGHT = FILTER_BATCH * FILTER_SIZE;
    const int dimension = std::ceil(std::sqrt(LENGHT));
    dim3 grid_size(cuDivUp(dimension, GRID_SIZE),
                   cuDivUp(dimension, GRID_SIZE));
    dim3 block_size(GRID_SIZE, GRID_SIZE);

    const int BYTE = LENGHT * sizeof(hipfftComplex);
    hipfftComplex *d_compl_out;
    hipMalloc(reinterpret_cast<void**>(&d_compl_out), BYTE);
    
    invConjuateConvKernel<<<grid_size, block_size>>>(d_compl_out, d_complex,
                                                     d_compl_model, LENGHT);
    
    return d_compl_out;
}

