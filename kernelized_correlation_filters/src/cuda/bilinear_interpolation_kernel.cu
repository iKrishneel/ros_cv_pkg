#include "hip/hip_runtime.h"

#include <kernelized_correlation_filters/bilinear_interpolation_kernel.h>

__global__
void bilinearInterpolationKernel(float * d_result,
                                 const float *d_data,
                                 const int nx, const int ny,
                                 const int num_filters,  //! 256
                                 const caffeFilterInfo blob_info) {

    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;
    offset *= num_filters;  // ????
    if (offset < blob_info.data_lenght) {
       
       const float fx = static_cast<float>(blob_info.width)/
          static_cast<float>(nx);
       const float fy = static_cast<float>(blob_info.height)/
          static_cast<float>(ny);
       
       //! indvidual for loops
       for (int j = 0; j < ny; j++) {
          for (int i = 0; i < nx; i++) {
             float src_x = i * fx;
             float src_y = j * fy;
             
             int x1 = static_cast<int>(floorf(src_x));
             int y1 = static_cast<int>(floorf(src_y));


             float p1 = d_data[x1 + (y1 * blob_info.width)];
             float p2 = d_data[x1 + 1 + (y1 * blob_info.width)];
             float p3 = d_data[x1 + ((y1 + 1) * blob_info.width)];
             float p4 = d_data[x1 + 1+ ((y1 + 1)* blob_info.width)];
             
             if (i == 10 && j == 10) {
                printf("%d ", x1 + (y1 * blob_info.width));
                printf("%d ", x1 + 1 + (y1 * blob_info.width));
                printf("%d ", x1 + ((y1 + 1) * blob_info.width));
                printf("%d \n", x1 + 1 + ((y1 + 1) * blob_info.width));
                printf("%d \n", blob_info.width);
             }
             
             float wx = i - x1;
             float wy = j - y1;
             float wx1 = 1.0f - wx;
             float wy1 = 1.0f - wy;

             int w1 = wx1 * wy1 * 255.0f;
             int w2 = wx * wy1 * 255.0f;
             int w3 = wx1 * wy * 255.0f;
             int w4 = wx * wy * 255.0f;

             // float out_value = p1 * w1 + p2 * w2 + p3 * w3 + p4 * w4;
             float out_value = (p1 + p2 + p3 + p4)/ 4;
             d_result[offset + i + (j * nx)] = out_value;
          }
       }
    }
}

float *bilinearInterpolationGPU(const float *d_data,
                                const int new_x, const int new_y,
                                const int fwidth, const int fheight,
                                const int flenght,
                                const int num_filters) {
    caffeFilterInfo cfinfo(fwidth, fheight, 1, flenght);
    
    const int dimension = std::ceil(std::sqrt(num_filters));
    dim3 block_size(cuDivUp(dimension, GRID_SIZE),
                    cuDivUp(dimension, GRID_SIZE));
    dim3 grid_size(GRID_SIZE, GRID_SIZE);

    int OUT_BYTE = sizeof(float) * new_y * new_x * num_filters;
    float *d_output;
    hipMalloc(reinterpret_cast<void**>(&d_output), OUT_BYTE);

    bilinearInterpolationKernel<<<block_size, grid_size>>>(
       d_output, d_data, new_x, new_y, num_filters, cfinfo);

    float *cpu_out = (float*)malloc(OUT_BYTE);
    hipMemcpy(cpu_out, d_output, OUT_BYTE, hipMemcpyDeviceToHost);

    for (int i = 0; i < new_y; i++) {
       for (int j = 0; j < new_x; j++) {
          printf("%3.5f ", cpu_out[j + i * new_x]);
       }
       printf("\n");

    }

    printf("SIZE: %d  %d\n", new_x, new_y);
    
    return d_output;
}


float *bilinear_test(float *data, const int in_byte) {

    float *d_data;
    hipMalloc(reinterpret_cast<void**>(&d_data), in_byte);
    hipMemcpy(d_data, data, in_byte, hipMemcpyHostToDevice);

    int new_x = 640;
    int new_y = 480;
    caffeFilterInfo cfinfo(320, 240, 1, 320 * 240);

    int OUT_BYTE = sizeof(float) * new_y * new_x * 1;
    float *d_output;
    hipMalloc(reinterpret_cast<void**>(&d_output), OUT_BYTE);
    
    bilinearInterpolationKernel<<<1, 1>>>(
       d_output, d_data, new_x, new_y, 1, cfinfo);

    float *cpu_out = (float*)malloc(OUT_BYTE);
    hipMemcpy(cpu_out, d_output, OUT_BYTE, hipMemcpyDeviceToHost);

    hipFree(d_output);
    hipFree(d_data);
    
    return cpu_out;
}

